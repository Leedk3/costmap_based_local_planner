#include "hip/hip_runtime.h"
#include "collision_check_with_path/collision_check.cuh"

#define PI 3.1415926535897
#define arraySize 100
__global__ void print_cuda_kernel_test() 
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}
// Run on GPU
__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

__global__ void calculate_collision_cuda_kernel(GridMap* device_grid_map)//PathCandidates* device_candidates)//, GridMap* device_grid_map, int grid_map_width_size, int grid_map_height_size)
{
    // int index = blockIdx.x * blockDim.x + threadIdx.x;
    // PathCandidates single_path = device_candidates[index];
    // device_candidates -> points_x_
    
    
    // GridMap * grid_map_ = (GridMap*)malloc(m_OccupancyGrid_ptr->info.width * sizeof(GridMap));
    // for (unsigned int width = 0; width < m_OccupancyGrid_ptr->info.width; width++){
    //     grid_map_[width].occupied_intensity_ = (int*)malloc(m_OccupancyGrid_ptr->info.height * sizeof(int));
    //     for (unsigned int height = 0; height < m_OccupancyGrid_ptr->info.height; height++)
    //     {
    //     if(m_OccupancyGrid_ptr->data[height * m_OccupancyGrid_ptr->info.width + width] > 0)
    //     {
    //         *(grid_map_[width].occupied_intensity_ + height) = 255;
    //         geometry_msgs::Pose obstacle;
    //         obstacle.position.x = width * m_OccupancyGrid_ptr->info.resolution + m_OccupancyGrid_ptr->info.resolution /2 + m_OccupancyGrid_ptr->info.origin.position.x;
    //         obstacle.position.y = height * m_OccupancyGrid_ptr->info.resolution + m_OccupancyGrid_ptr->info.resolution /2 + m_OccupancyGrid_ptr->info.origin.position.y;
    //         m_Obstacles.push_back(obstacle);
    //     }
    //     else
    //     {
    //         *(grid_map_[width].occupied_intensity_ + height) = 0;
    //     }
    //     }
    // }
    

    // for (int k =0; k < m_Obstacles.size(); k++)
    // {
    //     for (int i =0; i < RollOut.size(); i++)
    //     {
    //     for (int j = 0; j < RollOut.at(i).size(); j++)
    //     {
    //         double x_wpt_ = RollOut.at(i).at(j).pos.x;
    //         double y_wpt_ = RollOut.at(i).at(j).pos.y;      
    //         double dist_to_obstacle = sqrt(pow(m_Obstacles.at(k).position.x - x_wpt_ ,2) + pow(m_Obstacles.at(k).position.y - y_wpt_,2));
    //         if (dist_to_obstacle < m_obstacle_radius)
    //         {
    //         int once;
    //         if (once == k)
    //         continue;
    //         std::cout << "x: "<<m_Obstacles.at(k).position.x << " y: " << m_Obstacles.at(k).position.y << "dist: "<<  dist_to_obstacle<< std::endl;
    //         visualization_msgs::Marker test_marker;
    //         once = k;
    //         }  
    //     }
    //     }
    // }
}
__global__ void addKernel( int *c, const int *a, const int *b )
{
    // int i = threadIdx.x;
    int i = blockIdx.x ;
	if( i < arraySize )
		c[i] = a[i] + b[i];
}
void path_candidates_initialize(
    PathCandidates* candidates, int candidates_size,
    GridMap* grid_map, int grid_map_width_size, int grid_map_height_size)
{
    
    int a[arraySize];
    int b[arraySize];
    int c[arraySize];

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

	// fill the arrays 'a' and 'b' on the CPU
    for( int i = 0 ; i < arraySize ; i++ ) {
		a[i] = i;
		b[i] = i;
	}

	// Add vectors in parallel.
	// Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c, arraySize * sizeof(int));
    hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    hipMalloc((void**)&dev_b, arraySize * sizeof(int));

// copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<arraySize, 1>>>(dev_c, dev_a, dev_b);
    hipDeviceSynchronize();

// copy the array 'c' back from the GPU to the CPU
    hipMemcpy(&c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

// display the results
    for( int i = 0 ; i < arraySize ; i++ ) {
	printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	}

	// free the memory allocated on the GPU
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);



    // printf("initialize is working, candidates size: %d\n", candidates_size);
    // PathCandidates* device_candidates;
    // hipMalloc((void**)&device_candidates, candidates_size * sizeof(PathCandidates));
    // hipMemcpy(device_candidates, candidates, candidates_size * sizeof(PathCandidates), hipMemcpyHostToDevice);
    // printf("1\n");
    // for (int i =0; i < candidates_size; i++)
    // {
    //     printf("points size: %d \n" ,candidates[i].points_size_); 
    //     PathCandidates single_path = candidates[i];
    //     // points_x_
    //     hipMalloc((void**)&(device_candidates[i].points_x_), single_path.points_size_ * sizeof(double));
    //     hipMemcpy(device_candidates[i].points_x_, single_path.points_x_, single_path.points_size_ * sizeof(double), hipMemcpyHostToDevice);        
    //     //points_y_
    //     hipMalloc((void**)&(device_candidates[i].points_y_), single_path.points_size_ * sizeof(double));
    //     hipMemcpy(device_candidates[i].points_y_, single_path.points_y_, single_path.points_size_ * sizeof(double), hipMemcpyHostToDevice);
    // }

    // printf("grid size: (%d * %d) \n", grid_map_width_size, grid_map_height_size);
    // GridMap* device_grid_map;
    // hipMalloc((void**)&device_grid_map, grid_map_width_size * sizeof(GridMap));
    // hipMemcpy(device_grid_map, grid_map, grid_map_width_size * sizeof(GridMap), hipMemcpyHostToDevice);

    // for (int i =0; i < grid_map_width_size; i++){
    //     GridMap host_grid_map = grid_map[i];
    //     //host_grid_map_x
    //     hipMalloc((void**)&(device_grid_map[i].grid_x_), grid_map_height_size * sizeof(double));
    //     hipMemcpy(device_grid_map[i].grid_x_, host_grid_map.grid_x_, grid_map_height_size * sizeof(double), hipMemcpyHostToDevice);
    //     //host_grid_map_y
    //     hipMalloc((void**)&(device_grid_map[i].grid_y_), grid_map_height_size * sizeof(double));
    //     hipMemcpy(device_grid_map[i].grid_y_, host_grid_map.grid_y_, grid_map_height_size * sizeof(double), hipMemcpyHostToDevice);
    //     //intensity
    //     hipMalloc((void**)&(device_grid_map[i].occupied_intensity_), grid_map_height_size * sizeof(int));
    //     hipMemcpy(device_grid_map[i].occupied_intensity_, host_grid_map.occupied_intensity_, grid_map_height_size * sizeof(int), hipMemcpyHostToDevice);
    // }

    // dim3 grid(grid_map_width_size, grid_map_height_size);
    // // dim3 block(candidates_size, 100, 100);
  
    // // calculate_collision_cuda_kernel<<<grid, 1>>>(device_grid_map);

    // // calculate_collision_cuda_kernel<<<1, 1>>>();//, candidates_size); //, device_grid_map, grid_map_width_size, grid_map_height_size);

    // for (int i =0; i < candidates_size; i++){

    //     //points_x_
    //     hipMemcpy(candidates[i].points_x_, device_candidates[i].points_x_, candidates[i].points_size_ * sizeof(double), hipMemcpyDeviceToHost);
    //     //points_y_
    //     hipMemcpy(candidates[i].points_y_, device_candidates[i].points_y_, candidates[i].points_size_ * sizeof(double), hipMemcpyDeviceToHost);
    // }

    // for (int i =0; i < grid_map_width_size; i++)
    // {
    //     //host_grid_map_x
    //     hipMemcpy(grid_map[i].grid_x_, device_grid_map[i].grid_x_, grid_map[i].grid_x_size_ * sizeof(unsigned int), hipMemcpyDeviceToHost);
    //     //host_grid_map_y
    //     hipMemcpy(grid_map[i].grid_y_, device_grid_map[i].grid_y_, grid_map[i].grid_y_size_ * sizeof(unsigned int), hipMemcpyDeviceToHost);
    //     //intensity
    //     hipMemcpy(grid_map[i].occupied_intensity_, device_grid_map[i].occupied_intensity_, grid_map[i].occupied_intensity_size_ * sizeof(unsigned int), hipMemcpyDeviceToHost);
    // }

    // // Cleanup
    // printf("1\n");
    // for (int i =0; i < candidates_size; i++)
    // {
    //   free(candidates[i].points_x_); //    candidates[i].points_x_ = (double*)malloc(device_single_path.points_size_ * sizeof(double));
    //   free(candidates[i].points_y_); // device_single_path.points_y_ = (double*)malloc(device_single_path.points_size_ * sizeof(double));    
    // }
    // free(candidates);

    // printf("2\n");
    // for (int i =0; i < grid_map_width_size; i++)
    // {
    //   free(grid_map[i].grid_x_); 
    //   free(grid_map[i].grid_y_); 
    //   free(grid_map[i].occupied_intensity_);

    // }
    // free(grid_map);

    // printf("3\n");
    // for (int i =0; i < candidates_size; i++)
    // {
    //     hipFree(device_candidates[i].points_x_);
    //     hipFree(device_candidates[i].points_y_);
    // }
    // hipFree(device_candidates);
    // for (int i =0; i < candidates_size; i++)
    // {
    //     hipFree(device_grid_map[i].grid_x_);
    //     hipFree(device_grid_map[i].grid_y_);
    //     hipFree(device_grid_map[i].occupied_intensity_);
    // }
    // hipFree(device_grid_map);
}